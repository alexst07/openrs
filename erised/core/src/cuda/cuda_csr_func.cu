#include "hip/hip_runtime.h"
#include "cuda_csr_func.h"

namespace erised { namespace cuda {

template<class MapFn, class MapFn>
__global__
void GpuCsr<T>::CuMap(T* elems, size_t len, MapFn fn) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < len) {
    elems[idx] = fn(elems[idx]);
  }
}

}}
