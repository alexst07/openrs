#include "hip/hip_runtime.h"
#include "cuda_csr.h"

namespace erised { namespace cuda {

template<typename T>
template<typename MapFn>
__global__
void GpuCsr<T>::CuMap(MapFn&& fn) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < num_elems_) {
    elems_[idx] = fn(elems_[idx]);
  }
}

}}
